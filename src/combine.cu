#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
  float* out,
  const int* out_shape,
  const int* out_strides,
  float* a_storage,
  const int* a_shape,
  const int* a_strides,
  float* b_storage,
  const int* b_shape,
  const int* b_strides
) {

  __shared__ float a_shared[TILE][TILE];
  __shared__ float b_shared[TILE][TILE];

  int bx = blockIdx.x, by = blockIdx.y;
  int tx = threadIdx.x, ty = threadIdx.y;

  int batch = blockIdx.z;
  int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
  int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;

  int row = bx * blockDim.x + tx;
  int col = by * blockDim.y + ty;

  float pvalue = 0;

  for (int i = 0; i*TILE < a_shape[2]; ++i) {
      int a_col = i * TILE + ty;
      if(row < a_shape[1] && a_col < a_shape[2]) {
          a_shared[tx][ty] = a_storage[batch * a_batch_stride + row * a_strides[1] + a_col * a_strides[2]];
      } else {
          a_shared[tx][ty] = 0.0;
      }

      int b_row = i * TILE + tx;
      if(b_row < b_shape[1] && col < b_shape[2]) {
          b_shared[tx][ty] = b_storage[batch * b_batch_stride + b_row * b_strides[1] + col * b_strides[2]];
      } else {
          b_shared[tx][ty] = 0.0;
      }

      __syncthreads(); 

      for (int k = 0; k < TILE; ++k) {
        if ((i * TILE + k) < a_shape[2] && (i * TILE + k) < b_shape[1])
          pvalue += a_shared[tx][k] * b_shared[k][ty];
      }

      __syncthreads(); 
  }

  if (row < out_shape[1] && col < out_shape[2]) 
      out[batch * out_strides[0] + row * out_strides[1] + col * out_strides[2]] = pvalue;
}


__global__ void mapKernel(
  float* out, 
  int* out_shape, 
  int* out_strides, 
  int out_size, 
  float* in_storage, 
  int* in_shape, 
  int* in_strides,
  int shape_size,
  int fn_id
) {
/**
 * Map function. Apply a unary function to each element of the input array and store the result in the output array.
 * Optimization: Parallelize over the elements of the output array.
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 * - broadcast_index: converts an index in a smaller array to an index in a larger array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  in_storage: compact 1D array of size in_size
 *  in_shape: shape of the input array
 *  in_strides: strides of the input array
 *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
 *  fn_id: id of the function to apply to each element of the input array
 *
 * Returns:
 *  None (Fills in out array)
 */

  int out_index[MAX_DIMS];
  int in_index[MAX_DIMS];
  
  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the position in the output array that this thread will write to
  // 2. Convert the position to the out_index according to out_shape
  // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
  // 4. Calculate the position of element in in_array according to in_index and in_strides
  // 5. Calculate the position of element in out_array according to out_index and out_strides
  // 6. Apply the unary function to the input element and write the output to the out memory
  

  //Task 1
  int position = blockIdx.x*blockDim.x + threadIdx.x;
  if (position >= out_size) return;

  to_index(position, out_shape, out_index, shape_size);
  //Task 3
  broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
  //Task 4
  int in_position = index_to_position(in_index, in_strides, shape_size);
  //Task 5
  // int out_position = index_to_position(out_index, out_strides, shape_size);
  //Task 6
  out[position] = fn(fn_id, in_storage[in_position]);
  /// END ASSIGN1_2
}


__global__ void reduceKernel(
  float* out,
  int* out_shape,
  int* out_strides,
  int out_size,
  float* a_storage,
  int* a_shape,
  int* a_strides,
  int reduce_dim,
  float reduce_value,
  int shape_size,
  int fn_id
) {
/**
 * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
 * Optimization:
 * Parallelize over the reduction operation. Each kernel performs one reduction.
 * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  a_storage: compact 1D array of size in_size
 *  a_shape: shape of the input array
 *  a_strides: strides of the input array
 *  reduce_dim: dimension to reduce on
 *  reduce_value: initial value for the reduction
 *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
 *  fn_id: id of the reduce function, currently only support add, multiply, and max
 *
 *
 * Returns:
 *  None (Fills in out array)
 */

  // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
  int out_index[MAX_DIMS];

  /// BEGIN ASSIGN1_2
  /// TODO
  // 1. Define the position of the output element that this thread or this block will write to
  int out_pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (out_pos >= out_size) return;

  // 2. Convert the out_pos to the out_index according to out_shape
  to_index(out_pos, out_shape, out_index, shape_size);

  // 3. Initialize the reduce_value to the output element
  float result = reduce_value;
  //Check this
  // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
  for (int i = 0; i < a_shape[reduce_dim]; ++i) {
      out_index[reduce_dim] = i;
      int a_position = index_to_position(out_index, a_strides, shape_size);
      result = fn(fn_id, result, a_storage[a_position]);
  }

  // 5. Write the reduced value to out memory
  out[out_pos] = result;
  /// END ASSIGN1_2
}

__global__ void zipKernel(
  float* out,
  int* out_shape,
  int* out_strides,
  int out_size,
  int out_shape_size,
  float* a_storage,
  int* a_shape,
  int* a_strides,
  int a_shape_size,
  float* b_storage, 
  int* b_shape, 
  int* b_strides,
  int b_shape_size,
  int fn_id
) {
/**
 * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
 * Optimization: Parallelize over the elements of the output array.
 *
 * You may find the following functions useful:
 * - index_to_position: converts an index to a position in a compact array
 * - to_index: converts a position to an index in a multidimensional array
 * - broadcast_index: converts an index in a smaller array to an index in a larger array
 *
 * Args:
 *  out: compact 1D array of size out_size to write the output to
 *  out_shape: shape of the output array
 *  out_strides: strides of the output array
 *  out_size: size of the output array
 *  out_shape_size: number of dimensions in the output array
 *  a_storage: compact 1D array of size in_size
 *  a_shape: shape of the input array
 *  a_strides: strides of the input array
 *  a_shape_size: number of dimensions in the input array
 *  b_storage: compact 1D array of size in_size
 *  b_shape: shape of the input array
 *  b_strides: strides of the input array
 *  b_shape_size: number of dimensions in the input array
 *  fn_id: id of the function to apply to each element of the a & b array
 *
 *
 * Returns:
 *  None (Fills in out array)
 */

  int out_index[MAX_DIMS];
  int a_index[MAX_DIMS];
  int b_index[MAX_DIMS];
  //Zip function
  /// BEGIN ASSIGN1_2
  /// TODO
  // Hints:
  // 1. Compute the position in the output array that this thread will write to
  // 2. Convert the position to the out_index according to out_shape
  // 3. Calculate the position of element in out_array according to out_index and out_strides
  // 4. Broadcast the out_index to the a_index according to a_shape
  // 5. Calculate the position of element in a_array according to a_index and a_strides
  // 6. Broadcast the out_index to the b_index according to b_shape
  // 7.Calculate the position of element in b_array according to b_index and b_strides
  // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
  
  //Task1
  int thread_pos = blockIdx.x*blockDim.x + threadIdx.x;
  if (thread_pos>=out_size) return;
  //Task2
  to_index(thread_pos, out_shape, out_index, out_shape_size);
  //Task3
  int out_pos = index_to_position(out_index, out_strides, out_shape_size);
  //Task4
  broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
  //Task5
  int a_pos = index_to_position(a_index, a_strides, a_shape_size);
  //Task6
  broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
  //Task7
  int b_pos = index_to_position(b_index, b_strides, b_shape_size);
  //Task8
  out[out_pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);
  //assert(false && "Not Implemented");
  /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}